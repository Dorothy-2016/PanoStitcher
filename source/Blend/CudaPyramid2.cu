#include "hip/hip_runtime.h"
#include "CudaUtil.cuh"
#include <opencv2/core/core.hpp>
#include <opencv2/core/cuda_devptrs.hpp>
#include <opencv2/gpu/gpu.hpp>
#include <opencv2/gpu/stream_accessor.hpp>
#include <opencv2/gpu/device/common.hpp>
//#include <opencv2/gpu/device/border_interpolate.hpp>
//#include <opencv2/gpu/device/vec_traits.hpp>
//#include <opencv2/gpu/device/vec_math.hpp>
//#include <opencv2/gpu/device/saturate_cast.hpp>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>

#define PYR_DOWN_BLOCK_SIZE 256
#define PYR_UP_BLOCK_WIDTH 16
#define PYR_UP_BLOCK_HEIGHT 16
#define UTIL_BLOCK_WIDTH 32
#define UTIL_BLOCK_HEIGHT 8

/*struct BrdRowReflect101
{
    explicit __host__ __device__ __forceinline__ BrdRowReflect101(int width) : last_col(width - 1) {}

    __device__ __forceinline__ int idx_col_low(int x) const
    {
        return ::abs(x) % (last_col + 1);
    }

    __device__ __forceinline__ int idx_col_high(int x) const
    {
        return ::abs(last_col - ::abs(last_col - x)) % (last_col + 1);
    }

    __device__ __forceinline__ int idx_col(int x) const
    {
        return idx_col_low(idx_col_high(x));
    }

    const int last_col;
};

struct BrdColReflect101
{
    explicit __host__ __device__ __forceinline__ BrdColReflect101(int height) : last_row(height - 1) {}

    __device__ __forceinline__ int idx_row_low(int y) const
    {
        return ::abs(y) % (last_row + 1);
    }

    __device__ __forceinline__ int idx_row_high(int y) const
    {
        return ::abs(last_row - ::abs(last_row - y)) % (last_row + 1);
    }

    __device__ __forceinline__ int idx_row(int y) const
    {
        return idx_row_low(idx_row_high(y));
    }

    const int last_row;
};

struct BrdRowWrap
{
    explicit __host__ __device__ __forceinline__ BrdRowWrap(int width_) : width(width_) {}

    __device__ __forceinline__ int idx_col_low(int x) const
    {
        //return (x >= 0) * x + (x < 0) * (x - ((x - width + 1) / width) * width);
        if (x >= 0) return x;
        else return (x < 0) * (x - ((x - width + 1) / width) * width);
    }

    __device__ __forceinline__ int idx_col_high(int x) const
    {
        //return (x < width) * x + (x >= width) * (x % width);
        if (x < width) return x;
        else return (x % width);
    }

    __device__ __forceinline__ int idx_col(int x) const
    {
        return idx_col_high(idx_col_low(x));
    }

    const int width;
};

struct BrdColWrap
{
    explicit __host__ __device__ __forceinline__ BrdColWrap(int height_) : height(height_) {}

    __device__ __forceinline__ int idx_row_low(int y) const
    {
        //return (y >= 0) * y + (y < 0) * (y - ((y - height + 1) / height) * height);
        if (y >= 0) return y;
        else return (y - ((y - height + 1) / height) * height);
    }

    __device__ __forceinline__ int idx_row_high(int y) const
    {
        //return (y < height) * y + (y >= height) * (y % height);
        if (y < height) return y;
        else return (y % height);
    }

    __device__ __forceinline__ int idx_row(int y) const
    {
        return idx_row_high(idx_row_low(y));
    }

    const int height;
};

template<typename Type>
__device__ __forceinline__ Type getElem(const unsigned char* data, int step, int row, int col)
{
    return *((Type*)(data + row * step) + col);
}

template<typename Type>
__device__ __forceinline__ Type getElem(unsigned char* data, int step, int row, int col)
{
    return *((Type*)(data + row * step) + col);
}

template<typename Type>
__device__ __forceinline__ const Type* getRowPtr(const unsigned char* data, int step, int row)
{
    return (const Type*)(data + row * step);
}

template<typename Type>
__device__ __forceinline__ Type* getRowPtr(unsigned char* data, int step, int row)
{
    return (Type*)(data + row * step);
}

__device__ __forceinline__ int4 operator*(int scale, short4 val)
{
    int4 ret;
    ret.x = scale * val.x;
    ret.y = scale * val.y;
    ret.z = scale * val.z;
    return ret;
}

__device__ __forceinline__ int4 operator+(int4 a, int4 b)
{
    int4 ret;
    ret.x = a.x + b.x;
    ret.y = a.y + b.y;
    ret.z = a.z + b.z;
    return ret;
}

__device__ __forceinline__ int4 operator-(int4 a, int4 b)
{
    int4 ret;
    ret.x = a.x - b.x;
    ret.y = a.y - b.y;
    ret.z = a.z - b.z;
    return ret;
}

__device__ __forceinline__ short4 operator-(short4 a, short4 b)
{
    short4 ret;
    ret.x = a.x - b.x;
    ret.y = a.y - b.y;
    ret.z = a.z - b.z;
    return ret;
}

__device__ __forceinline__ int4 operator*(short scale, int4 val)
{
    int4 ret;
    ret.x = scale * val.x;
    ret.y = scale * val.y;
    ret.z = scale * val.z;
    return ret;
}

__device__ __forceinline__ int4 operator*(int scale, int4 val)
{
    int4 ret;
    ret.x = scale * val.x;
    ret.y = scale * val.y;
    ret.z = scale * val.z;
    return ret;
}

__device__ __forceinline__ short4 operator/(int4 val, int scale)
{
    short4 ret;
    ret.x = val.x / scale;
    ret.y = val.y / scale;
    ret.z = val.z / scale;
    return ret;
}

__device__ __forceinline__ int4& operator+=(int4& a, int4& b)
{
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
    return a;
}

__device__ __forceinline__ int4 operator>>(int4 val, int amount)
{
    int4 ret;
    ret.x = val.x >> amount;
    ret.y = val.y >> amount;
    ret.z = val.z >> amount;
    return ret;
}

__device__ __forceinline__ int4 operator<<(int4 val, int amount)
{
    int4 ret;
    ret.x = val.x << amount;
    ret.y = val.y << amount;
    ret.z = val.z << amount;
    return ret;
}

__device__ __forceinline__ int4 roundCastShift6ToInt4(int4 vec)
{
    int4 ret;
    ret.x = (vec.x + 32) >> 6;
    ret.y = (vec.y + 32) >> 6;
    ret.z = (vec.z + 32) >> 6;
    return ret;
}

__device__ __forceinline__ short4 roundCastShift6ToShort4(int4 vec)
{
    short4 ret;
    ret.x = (vec.x + 32) >> 6;
    ret.y = (vec.y + 32) >> 6;
    ret.z = (vec.z + 32) >> 6;
    return ret;
}

__device__ __forceinline__ short4 roundCastShift8ToShort4(int4 vec)
{
    short4 ret;
    ret.x = (vec.x + 128) >> 8;
    ret.y = (vec.y + 128) >> 8;
    ret.z = (vec.z + 128) >> 8;
    return ret;
}*/

template<typename ColWiseReflectType, typename RowWiseReflectType>
__global__ void pyrDown16SC1To32SC1(const unsigned char* srcData, int srcRows, int srcCols, int srcStep,
    unsigned char* dstData, int dstRows, int dstCols, int dstStep, 
    const ColWiseReflectType rb, const RowWiseReflectType cb)
{
    __shared__ int smem[PYR_DOWN_BLOCK_SIZE + 4];

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y;

    const int srcy = 2 * y;

    if (srcy >= 2 && srcy < srcRows - 2 && x >= 2 && x < srcCols - 2)
    {
        {
            int sum;
            sum =           getElem<short>(srcData, srcStep, srcy - 2, x);
            sum = sum + 4 * getElem<short>(srcData, srcStep, srcy - 1, x);
            sum = sum + 6 * getElem<short>(srcData, srcStep, srcy,     x);
            sum = sum + 4 * getElem<short>(srcData, srcStep, srcy + 1, x);
            sum = sum +     getElem<short>(srcData, srcStep, srcy + 2, x);
            smem[2 + threadIdx.x] = sum;
        }

        if (threadIdx.x < 2)
        {
            const int leftx = x - 2;
            int sum;
            sum =           getElem<short>(srcData, srcStep, srcy - 2, leftx);
            sum = sum + 4 * getElem<short>(srcData, srcStep, srcy - 1, leftx);
            sum = sum + 6 * getElem<short>(srcData, srcStep, srcy,     leftx);
            sum = sum + 4 * getElem<short>(srcData, srcStep, srcy + 1, leftx);
            sum = sum +     getElem<short>(srcData, srcStep, srcy + 2, leftx);
            smem[threadIdx.x] = sum;
        }

        if (threadIdx.x > PYR_DOWN_BLOCK_SIZE - 3)
        {
            const int rightx = x + 2;
            int sum;
            sum =           getElem<short>(srcData, srcStep, srcy - 2, rightx);
            sum = sum + 4 * getElem<short>(srcData, srcStep, srcy - 1, rightx);
            sum = sum + 6 * getElem<short>(srcData, srcStep, srcy,     rightx);
            sum = sum + 4 * getElem<short>(srcData, srcStep, srcy + 1, rightx);
            sum = sum +     getElem<short>(srcData, srcStep, srcy + 2, rightx);
            smem[4 + threadIdx.x] = sum;
        }
    }
    else
    {
        {
            int sum;
            sum =           getElem<short>(srcData, srcStep, rb.idx_row_low(srcy - 2),  cb.idx_col_high(x));
            sum = sum + 4 * getElem<short>(srcData, srcStep, rb.idx_row_low(srcy - 1),  cb.idx_col_high(x));
            sum = sum + 6 * getElem<short>(srcData, srcStep, srcy,                      cb.idx_col_high(x));
            sum = sum + 4 * getElem<short>(srcData, srcStep, rb.idx_row_high(srcy + 1), cb.idx_col_high(x));
            sum = sum +     getElem<short>(srcData, srcStep, rb.idx_row_high(srcy + 2), cb.idx_col_high(x));
            smem[2 + threadIdx.x] = sum;
        }

        if (threadIdx.x < 2)
        {
            const int leftx = x - 2;
            int sum;
            sum =           getElem<short>(srcData, srcStep, rb.idx_row_low(srcy - 2),  cb.idx_col(leftx));
            sum = sum + 4 * getElem<short>(srcData, srcStep, rb.idx_row_low(srcy - 1),  cb.idx_col(leftx));
            sum = sum + 6 * getElem<short>(srcData, srcStep, srcy,                      cb.idx_col(leftx));
            sum = sum + 4 * getElem<short>(srcData, srcStep, rb.idx_row_high(srcy + 1), cb.idx_col(leftx));
            sum = sum +     getElem<short>(srcData, srcStep, rb.idx_row_high(srcy + 2), cb.idx_col(leftx));
            smem[threadIdx.x] = sum;
        }

        if (threadIdx.x > PYR_DOWN_BLOCK_SIZE - 3)
        {
            const int rightx = x + 2;
            int sum;
            sum =           getElem<short>(srcData, srcStep, rb.idx_row_low(srcy - 2),  cb.idx_col_high(rightx));
            sum = sum + 4 * getElem<short>(srcData, srcStep, rb.idx_row_low(srcy - 1),  cb.idx_col_high(rightx));
            sum = sum + 6 * getElem<short>(srcData, srcStep, srcy,                      cb.idx_col_high(rightx));
            sum = sum + 4 * getElem<short>(srcData, srcStep, rb.idx_row_high(srcy + 1), cb.idx_col_high(rightx));
            sum = sum +     getElem<short>(srcData, srcStep, rb.idx_row_high(srcy + 2), cb.idx_col_high(rightx));
            smem[4 + threadIdx.x] = sum;
        }
    }

    __syncthreads();

    //if (x == 0)
    //{
    //    printf("%d, %d, %d, %d, %d\n", smem[0], smem[1], smem[2], smem[3], smem[4]);
    //}
    if (threadIdx.x < PYR_DOWN_BLOCK_SIZE / 2)
    {
        const int tid2 = threadIdx.x * 2;
        int sum;
        sum =           smem[2 + tid2 - 2];
        sum = sum + 4 * smem[2 + tid2 - 1];
        sum = sum + 6 * smem[2 + tid2    ];
        sum = sum + 4 * smem[2 + tid2 + 1];
        sum = sum +     smem[2 + tid2 + 2];

        const int dstx = (blockIdx.x * blockDim.x + tid2) / 2;

        if (dstx < dstCols)
            getRowPtr<int>(dstData, dstStep, y)[dstx] = sum;
    }
}

template<typename ColWiseReflectType, typename RowWiseReflectType>
__global__ void pyrDown16SC1To16SC1(const unsigned char* srcData, int srcRows, int srcCols, int srcStep,
    unsigned char* dstData, int dstRows, int dstCols, int dstStep, 
    const ColWiseReflectType rb, const RowWiseReflectType cb)
{
    __shared__ int smem[PYR_DOWN_BLOCK_SIZE + 4];

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y;

    const int srcy = 2 * y;

    if (srcy >= 2 && srcy < srcRows - 2 && x >= 2 && x < srcCols - 2)
    {
        {
            int sum;
            sum =           getElem<short>(srcData, srcStep, srcy - 2, x);
            sum = sum + 4 * getElem<short>(srcData, srcStep, srcy - 1, x);
            sum = sum + 6 * getElem<short>(srcData, srcStep, srcy,     x);
            sum = sum + 4 * getElem<short>(srcData, srcStep, srcy + 1, x);
            sum = sum +     getElem<short>(srcData, srcStep, srcy + 2, x);
            smem[2 + threadIdx.x] = sum;
        }

        if (threadIdx.x < 2)
        {
            const int leftx = x - 2;
            int sum;
            sum =           getElem<short>(srcData, srcStep, srcy - 2, leftx);
            sum = sum + 4 * getElem<short>(srcData, srcStep, srcy - 1, leftx);
            sum = sum + 6 * getElem<short>(srcData, srcStep, srcy,     leftx);
            sum = sum + 4 * getElem<short>(srcData, srcStep, srcy + 1, leftx);
            sum = sum +     getElem<short>(srcData, srcStep, srcy + 2, leftx);
            smem[threadIdx.x] = sum;
        }

        if (threadIdx.x > PYR_DOWN_BLOCK_SIZE - 3)
        {
            const int rightx = x + 2;
            int sum;
            sum =           getElem<short>(srcData, srcStep, srcy - 2, rightx);
            sum = sum + 4 * getElem<short>(srcData, srcStep, srcy - 1, rightx);
            sum = sum + 6 * getElem<short>(srcData, srcStep, srcy,     rightx);
            sum = sum + 4 * getElem<short>(srcData, srcStep, srcy + 1, rightx);
            sum = sum +     getElem<short>(srcData, srcStep, srcy + 2, rightx);
            smem[4 + threadIdx.x] = sum;
        }
    }
    else
    {
        {
            int sum;
            sum =           getElem<short>(srcData, srcStep, rb.idx_row_low(srcy - 2),  cb.idx_col_high(x));
            sum = sum + 4 * getElem<short>(srcData, srcStep, rb.idx_row_low(srcy - 1),  cb.idx_col_high(x));
            sum = sum + 6 * getElem<short>(srcData, srcStep, srcy,                      cb.idx_col_high(x));
            sum = sum + 4 * getElem<short>(srcData, srcStep, rb.idx_row_high(srcy + 1), cb.idx_col_high(x));
            sum = sum +     getElem<short>(srcData, srcStep, rb.idx_row_high(srcy + 2), cb.idx_col_high(x));
            smem[2 + threadIdx.x] = sum;
        }

        if (threadIdx.x < 2)
        {
            const int leftx = x - 2;
            int sum;
            sum =           getElem<short>(srcData, srcStep, rb.idx_row_low(srcy - 2),  cb.idx_col(leftx));
            sum = sum + 4 * getElem<short>(srcData, srcStep, rb.idx_row_low(srcy - 1),  cb.idx_col(leftx));
            sum = sum + 6 * getElem<short>(srcData, srcStep, srcy,                      cb.idx_col(leftx));
            sum = sum + 4 * getElem<short>(srcData, srcStep, rb.idx_row_high(srcy + 1), cb.idx_col(leftx));
            sum = sum +     getElem<short>(srcData, srcStep, rb.idx_row_high(srcy + 2), cb.idx_col(leftx));
            smem[threadIdx.x] = sum;
        }

        if (threadIdx.x > PYR_DOWN_BLOCK_SIZE - 3)
        {
            const int rightx = x + 2;
            int sum;
            sum =           getElem<short>(srcData, srcStep, rb.idx_row_low(srcy - 2),  cb.idx_col_high(rightx));
            sum = sum + 4 * getElem<short>(srcData, srcStep, rb.idx_row_low(srcy - 1),  cb.idx_col_high(rightx));
            sum = sum + 6 * getElem<short>(srcData, srcStep, srcy,                      cb.idx_col_high(rightx));
            sum = sum + 4 * getElem<short>(srcData, srcStep, rb.idx_row_high(srcy + 1), cb.idx_col_high(rightx));
            sum = sum +     getElem<short>(srcData, srcStep, rb.idx_row_high(srcy + 2), cb.idx_col_high(rightx));
            smem[4 + threadIdx.x] = sum;
        }
    }

    __syncthreads();

    if (threadIdx.x < PYR_DOWN_BLOCK_SIZE / 2)
    {
        const int tid2 = threadIdx.x * 2;
        int sum;
        sum =           smem[2 + tid2 - 2];
        sum = sum + 4 * smem[2 + tid2 - 1];
        sum = sum + 6 * smem[2 + tid2    ];
        sum = sum + 4 * smem[2 + tid2 + 1];
        sum = sum +     smem[2 + tid2 + 2];

        const int dstx = (blockIdx.x * blockDim.x + tid2) / 2;

        if (dstx < dstCols)
            getRowPtr<short>(dstData, dstStep, y)[dstx] = (sum + 128) >> 8;
    }
}

template<typename ColWiseReflectType, typename RowWiseReflectType>
__global__ void pyrDown16SC4To32SC4(const unsigned char* srcData, int srcRows, int srcCols, int srcStep,
    unsigned char* dstData, int dstRows, int dstCols, int dstStep, 
    const ColWiseReflectType rb, const RowWiseReflectType cb)
{
    __shared__ int4 smem[PYR_DOWN_BLOCK_SIZE + 4];

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y;

    const int srcy = 2 * y;

    if (srcy >= 2 && srcy < srcRows - 2 && x >= 2 && x < srcCols - 2)
    {
        {
            int4 sum;
            sum =       1 * getElem<short4>(srcData, srcStep, srcy - 2, x);
            sum = sum + 4 * getElem<short4>(srcData, srcStep, srcy - 1, x);
            sum = sum + 6 * getElem<short4>(srcData, srcStep, srcy,     x);
            sum = sum + 4 * getElem<short4>(srcData, srcStep, srcy + 1, x);
            sum = sum + 1 * getElem<short4>(srcData, srcStep, srcy + 2, x);
            smem[2 + threadIdx.x] = sum;
        }

        if (threadIdx.x < 2)
        {
            const int leftx = x - 2;
            int4 sum;
            sum =       1 * getElem<short4>(srcData, srcStep, srcy - 2, leftx);
            sum = sum + 4 * getElem<short4>(srcData, srcStep, srcy - 1, leftx);
            sum = sum + 6 * getElem<short4>(srcData, srcStep, srcy,     leftx);
            sum = sum + 4 * getElem<short4>(srcData, srcStep, srcy + 1, leftx);
            sum = sum + 1 * getElem<short4>(srcData, srcStep, srcy + 2, leftx);
            smem[threadIdx.x] = sum;
        }

        if (threadIdx.x > PYR_DOWN_BLOCK_SIZE - 3)
        {
            const int rightx = x + 2;
            int4 sum;
            sum =       1 * getElem<short4>(srcData, srcStep, srcy - 2, rightx);
            sum = sum + 4 * getElem<short4>(srcData, srcStep, srcy - 1, rightx);
            sum = sum + 6 * getElem<short4>(srcData, srcStep, srcy,     rightx);
            sum = sum + 4 * getElem<short4>(srcData, srcStep, srcy + 1, rightx);
            sum = sum + 1 * getElem<short4>(srcData, srcStep, srcy + 2, rightx);
            smem[4 + threadIdx.x] = sum;
        }
    }
    else
    {
        {
            int4 sum;
            sum =       1 * getElem<short4>(srcData, srcStep, rb.idx_row_low(srcy - 2),  cb.idx_col_high(x));
            sum = sum + 4 * getElem<short4>(srcData, srcStep, rb.idx_row_low(srcy - 1),  cb.idx_col_high(x));
            sum = sum + 6 * getElem<short4>(srcData, srcStep, srcy,                      cb.idx_col_high(x));
            sum = sum + 4 * getElem<short4>(srcData, srcStep, rb.idx_row_high(srcy + 1), cb.idx_col_high(x));
            sum = sum + 1 * getElem<short4>(srcData, srcStep, rb.idx_row_high(srcy + 2), cb.idx_col_high(x));
            smem[2 + threadIdx.x] = sum;
        }

        if (threadIdx.x < 2)
        {
            const int leftx = x - 2;
            int4 sum;
            sum =       1 * getElem<short4>(srcData, srcStep, rb.idx_row_low(srcy - 2),  cb.idx_col(leftx));
            sum = sum + 4 * getElem<short4>(srcData, srcStep, rb.idx_row_low(srcy - 1),  cb.idx_col(leftx));
            sum = sum + 6 * getElem<short4>(srcData, srcStep, srcy,                      cb.idx_col(leftx));
            sum = sum + 4 * getElem<short4>(srcData, srcStep, rb.idx_row_high(srcy + 1), cb.idx_col(leftx));
            sum = sum + 1 * getElem<short4>(srcData, srcStep, rb.idx_row_high(srcy + 2), cb.idx_col(leftx));
            smem[threadIdx.x] = sum;
        }

        if (threadIdx.x > PYR_DOWN_BLOCK_SIZE - 3)
        {
            const int rightx = x + 2;
            int4 sum;
            sum =       1 * getElem<short4>(srcData, srcStep, rb.idx_row_low(srcy - 2),  cb.idx_col_high(rightx));
            sum = sum + 4 * getElem<short4>(srcData, srcStep, rb.idx_row_low(srcy - 1),  cb.idx_col_high(rightx));
            sum = sum + 6 * getElem<short4>(srcData, srcStep, srcy,                      cb.idx_col_high(rightx));
            sum = sum + 4 * getElem<short4>(srcData, srcStep, rb.idx_row_high(srcy + 1), cb.idx_col_high(rightx));
            sum = sum + 1 * getElem<short4>(srcData, srcStep, rb.idx_row_high(srcy + 2), cb.idx_col_high(rightx));
            smem[4 + threadIdx.x] = sum;
        }
    }

    __syncthreads();

    if (threadIdx.x < PYR_DOWN_BLOCK_SIZE / 2)
    {
        const int tid2 = threadIdx.x * 2;
        int4 sum;
        sum =       1 * smem[2 + tid2 - 2];
        sum = sum + 4 * smem[2 + tid2 - 1];
        sum = sum + 6 * smem[2 + tid2    ];
        sum = sum + 4 * smem[2 + tid2 + 1];
        sum = sum + 1 * smem[2 + tid2 + 2];

        const int dstx = (blockIdx.x * blockDim.x + tid2) / 2;

        if (dstx < dstCols)
            getRowPtr<int4>(dstData, dstStep, y)[dstx] = sum;
    }
}

template<typename ColWiseReflectType, typename RowWiseReflectType>
__global__ void pyrDown16SC4To16SC4(const unsigned char* srcData, int srcRows, int srcCols, int srcStep,
    unsigned char* dstData, int dstRows, int dstCols, int dstStep, 
    const ColWiseReflectType rb, const RowWiseReflectType cb)
{
    __shared__ int4 smem[PYR_DOWN_BLOCK_SIZE + 4];

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y;

    const int srcy = 2 * y;

    if (srcy >= 2 && srcy < srcRows - 2 && x >= 2 && x < srcCols - 2)
    {
        {
            int4 sum;
            sum =       1 * getElem<short4>(srcData, srcStep, srcy - 2, x);
            sum = sum + 4 * getElem<short4>(srcData, srcStep, srcy - 1, x);
            sum = sum + 6 * getElem<short4>(srcData, srcStep, srcy,     x);
            sum = sum + 4 * getElem<short4>(srcData, srcStep, srcy + 1, x);
            sum = sum + 1 * getElem<short4>(srcData, srcStep, srcy + 2, x);
            smem[2 + threadIdx.x] = sum;
        }

        if (threadIdx.x < 2)
        {
            const int leftx = x - 2;
            int4 sum;
            sum =       1 * getElem<short4>(srcData, srcStep, srcy - 2, leftx);
            sum = sum + 4 * getElem<short4>(srcData, srcStep, srcy - 1, leftx);
            sum = sum + 6 * getElem<short4>(srcData, srcStep, srcy,     leftx);
            sum = sum + 4 * getElem<short4>(srcData, srcStep, srcy + 1, leftx);
            sum = sum + 1 * getElem<short4>(srcData, srcStep, srcy + 2, leftx);
            smem[threadIdx.x] = sum;
        }

        if (threadIdx.x > PYR_DOWN_BLOCK_SIZE - 3)
        {
            const int rightx = x + 2;
            int4 sum;
            sum =       1 * getElem<short4>(srcData, srcStep, srcy - 2, rightx);
            sum = sum + 4 * getElem<short4>(srcData, srcStep, srcy - 1, rightx);
            sum = sum + 6 * getElem<short4>(srcData, srcStep, srcy,     rightx);
            sum = sum + 4 * getElem<short4>(srcData, srcStep, srcy + 1, rightx);
            sum = sum + 1 * getElem<short4>(srcData, srcStep, srcy + 2, rightx);
            smem[4 + threadIdx.x] = sum;
        }
    }
    else
    {
        {
            int4 sum;
            sum =       1 * getElem<short4>(srcData, srcStep, rb.idx_row_low(srcy - 2),  cb.idx_col_high(x));
            sum = sum + 4 * getElem<short4>(srcData, srcStep, rb.idx_row_low(srcy - 1),  cb.idx_col_high(x));
            sum = sum + 6 * getElem<short4>(srcData, srcStep, srcy,                      cb.idx_col_high(x));
            sum = sum + 4 * getElem<short4>(srcData, srcStep, rb.idx_row_high(srcy + 1), cb.idx_col_high(x));
            sum = sum + 1 * getElem<short4>(srcData, srcStep, rb.idx_row_high(srcy + 2), cb.idx_col_high(x));
            smem[2 + threadIdx.x] = sum;
        }

        if (threadIdx.x < 2)
        {
            const int leftx = x - 2;
            int4 sum;
            sum =       1 * getElem<short4>(srcData, srcStep, rb.idx_row_low(srcy - 2),  cb.idx_col(leftx));
            sum = sum + 4 * getElem<short4>(srcData, srcStep, rb.idx_row_low(srcy - 1),  cb.idx_col(leftx));
            sum = sum + 6 * getElem<short4>(srcData, srcStep, srcy,                      cb.idx_col(leftx));
            sum = sum + 4 * getElem<short4>(srcData, srcStep, rb.idx_row_high(srcy + 1), cb.idx_col(leftx));
            sum = sum + 1 * getElem<short4>(srcData, srcStep, rb.idx_row_high(srcy + 2), cb.idx_col(leftx));
            smem[threadIdx.x] = sum;
        }

        if (threadIdx.x > PYR_DOWN_BLOCK_SIZE - 3)
        {
            const int rightx = x + 2;
            int4 sum;
            sum =       1 * getElem<short4>(srcData, srcStep, rb.idx_row_low(srcy - 2),  cb.idx_col_high(rightx));
            sum = sum + 4 * getElem<short4>(srcData, srcStep, rb.idx_row_low(srcy - 1),  cb.idx_col_high(rightx));
            sum = sum + 6 * getElem<short4>(srcData, srcStep, srcy,                      cb.idx_col_high(rightx));
            sum = sum + 4 * getElem<short4>(srcData, srcStep, rb.idx_row_high(srcy + 1), cb.idx_col_high(rightx));
            sum = sum + 1 * getElem<short4>(srcData, srcStep, rb.idx_row_high(srcy + 2), cb.idx_col_high(rightx));
            smem[4 + threadIdx.x] = sum;
        }
    }

    __syncthreads();

    if (threadIdx.x < PYR_DOWN_BLOCK_SIZE / 2)
    {
        const int tid2 = threadIdx.x * 2;
        int4 sum;
        sum =       1 * smem[2 + tid2 - 2];
        sum = sum + 4 * smem[2 + tid2 - 1];
        sum = sum + 6 * smem[2 + tid2    ];
        sum = sum + 4 * smem[2 + tid2 + 1];
        sum = sum + 1 * smem[2 + tid2 + 2];

        const int dstx = (blockIdx.x * blockDim.x + tid2) / 2;

        if (dstx < dstCols)
            getRowPtr<short4>(dstData, dstStep, y)[dstx] = roundCastShift8ToShort4(sum);
    }
}

template<typename ColWiseReflectType, typename RowWiseReflectType>
__global__ void pyrUp32SC4To32SC4(const unsigned char* srcData, int srcRows, int srcCols, int srcStep,
    unsigned char* dstData, int dstRows, int dstCols, int dstStep, 
    const ColWiseReflectType rb, const RowWiseReflectType cb)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ int4 s_srcPatch[PYR_UP_BLOCK_HEIGHT / 2 + 2][PYR_UP_BLOCK_WIDTH / 2 + 2];
    __shared__ int4 s_dstPatch[PYR_UP_BLOCK_HEIGHT + 4][PYR_UP_BLOCK_WIDTH];

    if ((threadIdx.x < PYR_UP_BLOCK_WIDTH / 2 + 2) && (threadIdx.y < PYR_UP_BLOCK_HEIGHT / 2 + 2))
    {
        int srcx = static_cast<int>((blockIdx.x * blockDim.x) / 2 + threadIdx.x) - 1;
        int srcy = static_cast<int>((blockIdx.y * blockDim.y) / 2 + threadIdx.y) - 1;

        srcx = srcx < 0 ? cb.idx_col_low(srcx) : (srcx >= srcCols) ? cb.idx_col_high(srcx) : srcx;
        srcy = srcy < 0 ? rb.idx_row_low(srcy) : (srcy >= srcRows) ? rb.idx_row_high(srcy) : srcy;

        s_srcPatch[threadIdx.y][threadIdx.x] = getElem<int4>(srcData, srcStep, srcy, srcx);
    }

    __syncthreads();

    int4 sum = make_int4(0, 0, 0, 0);

    const int evenFlag = static_cast<int>((threadIdx.x & 1) == 0);
    const int oddFlag  = static_cast<int>((threadIdx.x & 1) != 0);
    const int eveny = ((threadIdx.y & 1) == 0);
    const int tidx = threadIdx.x;

    if (eveny)
    {
        sum =       (evenFlag    ) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx - 2) >> 1)];
        sum = sum + ( oddFlag * 4) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx - 1) >> 1)];
        sum = sum + (evenFlag * 6) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx    ) >> 1)];
        sum = sum + ( oddFlag * 4) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx + 1) >> 1)];
        sum = sum + (evenFlag    ) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx + 2) >> 1)];
    }

    s_dstPatch[2 + threadIdx.y][threadIdx.x] = sum;

    if (threadIdx.y < 2)
    {
        if (eveny)
        {
            sum =       (evenFlag    ) * s_srcPatch[0][1 + ((tidx - 2) >> 1)];
            sum = sum + ( oddFlag * 4) * s_srcPatch[0][1 + ((tidx - 1) >> 1)];
            sum = sum + (evenFlag * 6) * s_srcPatch[0][1 + ((tidx    ) >> 1)];
            sum = sum + ( oddFlag * 4) * s_srcPatch[0][1 + ((tidx + 1) >> 1)];
            sum = sum + (evenFlag    ) * s_srcPatch[0][1 + ((tidx + 2) >> 1)];
        }

        s_dstPatch[threadIdx.y][threadIdx.x] = sum;
    }

    if (threadIdx.y > PYR_UP_BLOCK_HEIGHT - 3)
    {
        if (eveny)
        {
            sum =       (evenFlag    ) * s_srcPatch[PYR_UP_BLOCK_HEIGHT / 2 + 1][1 + ((tidx - 2) >> 1)];
            sum = sum + ( oddFlag * 4) * s_srcPatch[PYR_UP_BLOCK_HEIGHT / 2 + 1][1 + ((tidx - 1) >> 1)];
            sum = sum + (evenFlag * 6) * s_srcPatch[PYR_UP_BLOCK_HEIGHT / 2 + 1][1 + ((tidx    ) >> 1)];
            sum = sum + ( oddFlag * 4) * s_srcPatch[PYR_UP_BLOCK_HEIGHT / 2 + 1][1 + ((tidx + 1) >> 1)];
            sum = sum + (evenFlag    ) * s_srcPatch[PYR_UP_BLOCK_HEIGHT / 2 + 1][1 + ((tidx + 2) >> 1)];
        }

        s_dstPatch[4 + threadIdx.y][threadIdx.x] = sum;
    }

    __syncthreads();

    const int tidy = threadIdx.y;

    sum =           s_dstPatch[2 + tidy - 2][threadIdx.x];
    sum = sum + 4 * s_dstPatch[2 + tidy - 1][threadIdx.x];
    sum = sum + 6 * s_dstPatch[2 + tidy    ][threadIdx.x];
    sum = sum + 4 * s_dstPatch[2 + tidy + 1][threadIdx.x];
    sum = sum +     s_dstPatch[2 + tidy + 2][threadIdx.x];

    if (x < dstCols && y < dstRows)
        getRowPtr<int4>(dstData, dstStep, y)[x] = roundCastShift6ToInt4(sum);
}

template<typename ColWiseReflectType, typename RowWiseReflectType>
__global__ void pyrUp16SC4To16SC4(const unsigned char* srcData, int srcRows, int srcCols, int srcStep,
    unsigned char* dstData, int dstRows, int dstCols, int dstStep, 
    const ColWiseReflectType rb, const RowWiseReflectType cb)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ short4 s_srcPatch[PYR_UP_BLOCK_HEIGHT / 2 + 2][PYR_UP_BLOCK_WIDTH / 2 + 2];
    __shared__ int4 s_dstPatch[PYR_UP_BLOCK_HEIGHT + 4][PYR_UP_BLOCK_WIDTH];

    if ((threadIdx.x < PYR_UP_BLOCK_WIDTH / 2 + 2) && (threadIdx.y < PYR_UP_BLOCK_HEIGHT / 2 + 2))
    {
        int srcx = static_cast<int>((blockIdx.x * blockDim.x) / 2 + threadIdx.x) - 1;
        int srcy = static_cast<int>((blockIdx.y * blockDim.y) / 2 + threadIdx.y) - 1;

        srcx = srcx < 0 ? cb.idx_col_low(srcx) : (srcx >= srcCols) ? cb.idx_col_high(srcx) : srcx;
        srcy = srcy < 0 ? rb.idx_row_low(srcy) : (srcy >= srcRows) ? rb.idx_row_high(srcy) : srcy;

        s_srcPatch[threadIdx.y][threadIdx.x] = getElem<short4>(srcData, srcStep, srcy, srcx);
    }

    __syncthreads();

    int4 sum = make_int4(0, 0, 0, 0);

    const int evenFlag = static_cast<int>((threadIdx.x & 1) == 0);
    const int oddFlag  = static_cast<int>((threadIdx.x & 1) != 0);
    const int eveny = ((threadIdx.y & 1) == 0);
    const int tidx = threadIdx.x;

    if (eveny)
    {
        sum =       (evenFlag    ) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx - 2) >> 1)];
        sum = sum + ( oddFlag * 4) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx - 1) >> 1)];
        sum = sum + (evenFlag * 6) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx    ) >> 1)];
        sum = sum + ( oddFlag * 4) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx + 1) >> 1)];
        sum = sum + (evenFlag    ) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx + 2) >> 1)];
    }

    s_dstPatch[2 + threadIdx.y][threadIdx.x] = sum;

    if (threadIdx.y < 2)
    {
        if (eveny)
        {
            sum =       (evenFlag    ) * s_srcPatch[0][1 + ((tidx - 2) >> 1)];
            sum = sum + ( oddFlag * 4) * s_srcPatch[0][1 + ((tidx - 1) >> 1)];
            sum = sum + (evenFlag * 6) * s_srcPatch[0][1 + ((tidx    ) >> 1)];
            sum = sum + ( oddFlag * 4) * s_srcPatch[0][1 + ((tidx + 1) >> 1)];
            sum = sum + (evenFlag    ) * s_srcPatch[0][1 + ((tidx + 2) >> 1)];
        }

        s_dstPatch[threadIdx.y][threadIdx.x] = sum;
    }

    if (threadIdx.y > PYR_UP_BLOCK_HEIGHT - 3)
    {
        if (eveny)
        {
            sum =       (evenFlag    ) * s_srcPatch[PYR_UP_BLOCK_HEIGHT / 2 + 1][1 + ((tidx - 2) >> 1)];
            sum = sum + ( oddFlag * 4) * s_srcPatch[PYR_UP_BLOCK_HEIGHT / 2 + 1][1 + ((tidx - 1) >> 1)];
            sum = sum + (evenFlag * 6) * s_srcPatch[PYR_UP_BLOCK_HEIGHT / 2 + 1][1 + ((tidx    ) >> 1)];
            sum = sum + ( oddFlag * 4) * s_srcPatch[PYR_UP_BLOCK_HEIGHT / 2 + 1][1 + ((tidx + 1) >> 1)];
            sum = sum + (evenFlag    ) * s_srcPatch[PYR_UP_BLOCK_HEIGHT / 2 + 1][1 + ((tidx + 2) >> 1)];
        }

        s_dstPatch[4 + threadIdx.y][threadIdx.x] = sum;
    }

    __syncthreads();

    const int tidy = threadIdx.y;

    sum =           s_dstPatch[2 + tidy - 2][threadIdx.x];
    sum = sum + 4 * s_dstPatch[2 + tidy - 1][threadIdx.x];
    sum = sum + 6 * s_dstPatch[2 + tidy    ][threadIdx.x];
    sum = sum + 4 * s_dstPatch[2 + tidy + 1][threadIdx.x];
    sum = sum +     s_dstPatch[2 + tidy + 2][threadIdx.x];

    if (x < dstCols && y < dstRows)
        getRowPtr<short4>(dstData, dstStep, y)[x] = roundCastShift6ToShort4(sum);
}

__global__ void divide32SC4To16SC4(const unsigned char* srcImageData, int srcImageRows, int srcImageCols, int srcImageStep,
    const unsigned char* srcAlphaData, int srcAlphaRows, int srcAlphaCols, int srcAlphaStep, 
    unsigned char* dstImageData, int dstImageRows, int dstImageCols, int dstImageStep,
    unsigned char* dstAlphaData, int dstAlphaRows, int dstAlphaCols, int dstAlphaStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < srcImageCols && y < srcImageRows)
    {
        int4 srcImageVal = getElem<int4>(srcImageData, srcImageStep, y, x);
        int srcAlphaVal = getElem<int>(srcAlphaData, srcAlphaStep, y, x);
        if (srcAlphaVal)
        {
            getRowPtr<short4>(dstImageData, dstImageStep, y)[x] = ((srcImageVal << 8) - srcImageVal) / srcAlphaVal;
            getRowPtr<short>(dstAlphaData, dstAlphaStep, y)[x] = 256;
        }
        else
        {
            getRowPtr<short4>(dstImageData, dstImageStep, y)[x] = make_short4(0, 0, 0, 0);
            getRowPtr<short>(dstAlphaData, dstAlphaStep, y)[x] = 0;
        }
    }
}

__global__ void accumulate16SC4To32SC4(const unsigned char* srcData, int srcRows, int srcCols, int srcStep,
    const unsigned char* weightData, int weightRows, int weightCols, int weightStep,
    unsigned char* dstData, int dstRows, int dstCols, int dstStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < srcCols && y < srcRows)
    {
        getRowPtr<int4>(dstData, dstStep, y)[x] = getRowPtr<int4>(dstData, dstStep, y)[x] + 
            getElem<short>(weightData, weightStep, y, x) * getElem<short4>(srcData, srcStep, y, x);
    }
}

__global__ void normalize32SC4(unsigned char* imageData, int imageRows, int imageCols, int imageStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < imageCols && y < imageRows)
    {
        getRowPtr<int4>(imageData, imageStep, y)[x] = (getElem<int4>(imageData, imageStep, y, x) + make_int4(128, 128, 128, 0)) >> 8;
    }
}

__global__ void scaledSet16SC1Mask16SC1(unsigned char* imageData, int imageRows, int imageCols, int imageStep,
    short val, const unsigned char* maskData, int maskStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < imageCols && y < imageRows)
    {
        getRowPtr<short>(imageData, imageStep, y)[x] = getElem<short>(maskData, maskStep, y, x) ? val : 0;
    }
}

__global__ void scaledSet16SC1Mask32SC1(unsigned char* imageData, int imageRows, int imageCols, int imageStep,
    short val, const unsigned char* maskData, int maskStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < imageCols && y < imageRows)
    {
        getRowPtr<short>(imageData, imageStep, y)[x] = getElem<int>(maskData, maskStep, y, x) ? val : 0;
    }
}

__global__ void divide32SC4To16SC4(const unsigned char* srcImageData, int srcImageRows, int srcImageCols, int srcImageStep,
    const unsigned char* srcAlphaData, int srcAlphaRows, int srcAlphaCols, int srcAlphaStep, 
    unsigned char* dstImageData, int dstImageRows, int dstImageCols, int dstImageStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < srcImageCols && y < srcImageRows)
    {
        int4 srcImageVal = getElem<int4>(srcImageData, srcImageStep, y, x);
        int srcAlphaVal = getElem<int>(srcAlphaData, srcAlphaStep, y, x);
        if (srcAlphaVal)
        {
            getRowPtr<short4>(dstImageData, dstImageStep, y)[x] = ((srcImageVal << 8) - srcImageVal) / srcAlphaVal;
        }
        else
        {
            getRowPtr<short4>(dstImageData, dstImageStep, y)[x] = make_short4(0, 0, 0, 0);
        }
    }
}

__global__ void subtract16SC4(const unsigned char* aData, int aStep, const unsigned char* bData, int bStep,
    unsigned char* cData, int cStep, int rows, int cols)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < cols && y < rows)
    {
        getRowPtr<short4>(cData, cStep, y)[x] = getElem<short4>(aData, aStep, y, x) - getElem<short4>(bData, bStep, y, x);
    }
}

__global__ void add32SC4(const unsigned char* aData, int aStep, const unsigned char* bData, int bStep,
    unsigned char* cData, int cStep, int rows, int cols)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < cols && y < rows)
    {
        getRowPtr<int4>(cData, cStep, y)[x] = getElem<int4>(aData, aStep, y, x) + getElem<int4>(bData, bStep, y, x);
    }
}

void pyramidDown16SC1To16SC1(const cv::gpu::GpuMat& src, cv::gpu::GpuMat& dst, cv::Size dstSize, bool horiWrap, cv::gpu::Stream& stream)
{
    CV_Assert(src.data && src.type() == CV_16SC1); 

    if (dstSize == cv::Size())
    {
        dstSize.width = ((src.cols + 1) >> 1);
        dstSize.height = ((src.rows + 1) >> 1);
    }
    dst.create(dstSize, CV_16SC1);

    hipStream_t st = cv::gpu::StreamAccessor::getStream(stream);
    const dim3 block(PYR_DOWN_BLOCK_SIZE);
    const dim3 grid(cv::gpu::divUp(src.cols, block.x), dst.rows);
    if (horiWrap)
    {
        BrdColReflect101 rb(src.rows);
        BrdRowWrap cb(src.cols);
        pyrDown16SC1To16SC1<<<grid, block, 0, st>>>(src.data, src.rows, src.cols, src.step, dst.data, dst.rows, dst.cols, dst.step, rb, cb);
    }
    else
    {
        BrdColReflect101 rb(src.rows);
        BrdRowReflect101 cb(src.cols);
        pyrDown16SC1To16SC1<<<grid, block, 0, st>>>(src.data, src.rows, src.cols, src.step, dst.data, dst.rows, dst.cols, dst.step, rb, cb);
    }
    //cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize());
}

void pyramidDown16SC1To32SC1(const cv::gpu::GpuMat& src, cv::gpu::GpuMat& dst, cv::Size dstSize, bool horiWrap, cv::gpu::Stream& stream)
{
    CV_Assert(src.data && src.type() == CV_16SC1); 

    if (dstSize == cv::Size())
    {
        dstSize.width = ((src.cols + 1) >> 1);
        dstSize.height = ((src.rows + 1) >> 1);
    }
    dst.create(dstSize, CV_32SC1);

    hipStream_t st = cv::gpu::StreamAccessor::getStream(stream);
    const dim3 block(PYR_DOWN_BLOCK_SIZE);
    const dim3 grid(cv::gpu::divUp(src.cols, block.x), dst.rows);
    if (horiWrap)
    {
        BrdColReflect101 rb(src.rows);
        BrdRowWrap cb(src.cols);
        pyrDown16SC1To32SC1 <<<grid, block, 0, st>>>(src.data, src.rows, src.cols, src.step, dst.data, dst.rows, dst.cols, dst.step, rb, cb);
    }
    else
    {
        BrdColReflect101 rb(src.rows);
        BrdRowReflect101 cb(src.cols);
        pyrDown16SC1To32SC1<<<grid, block, 0, st>>>(src.data, src.rows, src.cols, src.step, dst.data, dst.rows, dst.cols, dst.step, rb, cb);
    }
    //cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize());
}

void pyramidDown16SC4To32SC4(const cv::gpu::GpuMat& src, cv::gpu::GpuMat& dst, cv::Size dstSize, bool horiWrap, cv::gpu::Stream& stream)
{
    CV_Assert(src.data && src.type() == CV_16SC4); 

    if (dstSize == cv::Size())
    {
        dstSize.width = ((src.cols + 1) >> 1);
        dstSize.height = ((src.rows + 1) >> 1);
    }
    dst.create(dstSize, CV_32SC4);

    hipStream_t st = cv::gpu::StreamAccessor::getStream(stream);
    const dim3 block(PYR_DOWN_BLOCK_SIZE);
    const dim3 grid(cv::gpu::divUp(src.cols, block.x), dst.rows);
    if (horiWrap)
    {
        BrdColReflect101 rb(src.rows);
        BrdRowWrap cb(src.cols);
        pyrDown16SC4To32SC4<<<grid, block, 0, st>>>(src.data, src.rows, src.cols, src.step, dst.data, dst.rows, dst.cols, dst.step, rb, cb);
    }
    else
    {
        BrdColReflect101 rb(src.rows);
        BrdRowReflect101 cb(src.cols);
        pyrDown16SC4To32SC4<<<grid, block, 0, st>>>(src.data, src.rows, src.cols, src.step, dst.data, dst.rows, dst.cols, dst.step, rb, cb);
    }    
    //cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize());
}

void pyramidDown16SC4To16SC4(const cv::gpu::GpuMat& src, cv::gpu::GpuMat& dst, cv::Size dstSize, bool horiWrap, cv::gpu::Stream& stream)
{
    CV_Assert(src.data && src.type() == CV_16SC4); 

    if (dstSize == cv::Size())
    {
        dstSize.width = ((src.cols + 1) >> 1);
        dstSize.height = ((src.rows + 1) >> 1);
    }
    dst.create(dstSize, CV_16SC4);

    hipStream_t st = cv::gpu::StreamAccessor::getStream(stream);
    const dim3 block(PYR_DOWN_BLOCK_SIZE);
    const dim3 grid(cv::gpu::divUp(src.cols, block.x), dst.rows);
    if (horiWrap)
    {
        BrdColReflect101 rb(src.rows);
        BrdRowWrap cb(src.cols);
        pyrDown16SC4To16SC4<<<grid, block, 0, st>>>(src.data, src.rows, src.cols, src.step, dst.data, dst.rows, dst.cols, dst.step, rb, cb);
    }
    else
    {
        BrdColReflect101 rb(src.rows);
        BrdRowReflect101 cb(src.cols);
        pyrDown16SC4To16SC4<<<grid, block, 0, st>>>(src.data, src.rows, src.cols, src.step, dst.data, dst.rows, dst.cols, dst.step, rb, cb);
    }    
    //cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize());
}

void divide32SC4To16SC4(const cv::gpu::GpuMat& srcImage, const cv::gpu::GpuMat& srcAlpha,
    cv::gpu::GpuMat& dstImage, cv::gpu::GpuMat& dstAlpha, cv::gpu::Stream& stream)
{
    CV_Assert(srcImage.data && srcImage.type() == CV_32SC4 &&
        srcAlpha.data && srcAlpha.type() == CV_32SC1 &&
        srcImage.size() == srcAlpha.size());

    dstImage.create(srcImage.size(), CV_16SC4);
    dstAlpha.create(srcAlpha.size(), CV_16SC1);

    hipStream_t st = cv::gpu::StreamAccessor::getStream(stream);
    const dim3 block(UTIL_BLOCK_WIDTH, UTIL_BLOCK_HEIGHT);
    const dim3 grid(cv::gpu::divUp(srcImage.cols, block.x), cv::gpu::divUp(srcImage.rows, block.y));
    divide32SC4To16SC4<<<grid, block, 0, st>>>(srcImage.data, srcImage.rows, srcImage.cols, srcImage.step,
        srcAlpha.data, srcAlpha.rows, srcAlpha.cols, srcAlpha.step,
        dstImage.data, dstImage.rows, dstImage.cols, dstImage.step,
        dstAlpha.data, dstAlpha.rows, dstAlpha.cols, dstAlpha.step);
    //cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize());
}

void pyramidDown16SC4To16SC4(const cv::gpu::GpuMat& srcImage, const cv::gpu::GpuMat& srcAlpha, bool horiWrap, 
    cv::gpu::GpuMat& dstImage, cv::gpu::GpuMat& dstAlpha, cv::gpu::Stream& stream)
{
    CV_Assert(srcImage.data && srcImage.type() == CV_16SC4 &&
        srcAlpha.data && srcAlpha.type() == CV_16SC1 && srcImage.size() == srcAlpha.size());
    cv::gpu::GpuMat dstImage32S, dstAlpha32S;
    pyramidDown16SC4To32SC4(srcImage, dstImage32S, cv::Size(), horiWrap, stream);
    pyramidDown16SC1To32SC1(srcAlpha, dstAlpha32S, cv::Size(), horiWrap, stream);
    divide32SC4To16SC4(dstImage32S, dstAlpha32S, dstImage, dstAlpha, stream);
}

void pyramidUp32SC4To32SC4(const cv::gpu::GpuMat& src, cv::gpu::GpuMat& dst, cv::Size dstSize, bool horiWrap, cv::gpu::Stream& stream)
{
    CV_Assert(src.data && src.type() == CV_32SC4);
    
    if (dstSize == cv::Size())
    {
        dstSize.width = (src.cols << 1);
        dstSize.height = (src.rows << 1);
    }
    dst.create(dstSize, CV_32SC4);

    hipStream_t st = cv::gpu::StreamAccessor::getStream(stream);
    const dim3 block(PYR_UP_BLOCK_WIDTH, PYR_UP_BLOCK_HEIGHT);
    const dim3 grid(cv::gpu::divUp(dst.cols, block.x), cv::gpu::divUp(dst.rows, block.y));
    if (horiWrap)
    {
        BrdColReflect101 rb(src.rows);
        BrdRowWrap cb(src.cols);
        pyrUp32SC4To32SC4<<<grid, block, 0, st>>>(src.data, src.rows, src.cols, src.step, dst.data, dst.rows, dst.cols, dst.step, rb, cb);
    }
    else
    {
        BrdColReflect101 rb(src.rows);
        BrdRowReflect101 cb(src.cols);
        pyrUp32SC4To32SC4<<<grid, block, 0, st>>>(src.data, src.rows, src.cols, src.step, dst.data, dst.rows, dst.cols, dst.step, rb, cb);
    }
    //cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize());
}

void pyramidUp16SC4To16SC4(const cv::gpu::GpuMat& src, cv::gpu::GpuMat& dst, cv::Size dstSize, bool horiWrap, cv::gpu::Stream& stream)
{
    CV_Assert(src.data && src.type() == CV_16SC4);
    
    if (dstSize == cv::Size())
    {
        dstSize.width = (src.cols << 1);
        dstSize.height = (src.rows << 1);
    }
    dst.create(dstSize, CV_16SC4);

    hipStream_t st = cv::gpu::StreamAccessor::getStream(stream);
    const dim3 block(PYR_UP_BLOCK_WIDTH, PYR_UP_BLOCK_HEIGHT);
    const dim3 grid(cv::gpu::divUp(dst.cols, block.x), cv::gpu::divUp(dst.rows, block.y));
    if (horiWrap)
    {
        BrdColReflect101 rb(src.rows);
        BrdRowWrap cb(src.cols);
        pyrUp16SC4To16SC4<<<grid, block, 0, st>>>(src.data, src.rows, src.cols, src.step, dst.data, dst.rows, dst.cols, dst.step, rb, cb);
    }
    else
    {
        BrdColReflect101 rb(src.rows);
        BrdRowReflect101 cb(src.cols);
        pyrUp16SC4To16SC4<<<grid, block, 0, st>>>(src.data, src.rows, src.cols, src.step, dst.data, dst.rows, dst.cols, dst.step, rb, cb);
    }
    //cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize());
}

void accumulate16SC4To32SC4(const cv::gpu::GpuMat& src, const cv::gpu::GpuMat& weight, cv::gpu::GpuMat& dst, cv::gpu::Stream& stream)
{
    CV_Assert(src.data && src.type() == CV_16SC4 &&
        weight.data && weight.type() == CV_16SC1 &&
        dst.data && dst.type() == CV_32SC4 &&
        src.size() == weight.size() && src.size() == dst.size());

    hipStream_t st = cv::gpu::StreamAccessor::getStream(stream);
    const dim3 block(UTIL_BLOCK_WIDTH, UTIL_BLOCK_HEIGHT);
    const dim3 grid(cv::gpu::divUp(src.cols, block.x), cv::gpu::divUp(src.rows, block.y));
    accumulate16SC4To32SC4<<<grid, block, 0, st>>>(src.data, src.rows, src.cols, src.step, 
        weight.data, weight.rows, weight.cols, weight.step,
        dst.data, dst.rows, dst.cols, dst.step);
    //cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize());
}

void normalize32SC4(cv::gpu::GpuMat& image, cv::gpu::Stream& stream)
{
    CV_Assert(image.data && image.type() == CV_32SC4);
    hipStream_t st = cv::gpu::StreamAccessor::getStream(stream);
    const dim3 block(UTIL_BLOCK_WIDTH, UTIL_BLOCK_HEIGHT);
    const dim3 grid(cv::gpu::divUp(image.cols, block.x), cv::gpu::divUp(image.rows, block.y));
    normalize32SC4<<<grid, block, 0, st>>>(image.data, image.rows, image.cols, image.step);
    //cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize());
}

void scaledSet16SC1Mask16SC1(cv::gpu::GpuMat& image, short val, const cv::gpu::GpuMat& mask, cv::gpu::Stream& stream)
{
    CV_Assert(image.data && image.type() == CV_16SC1 &&
        mask.data && mask.type() == CV_16SC1 && image.size() == mask.size());
    hipStream_t st = cv::gpu::StreamAccessor::getStream(stream);
    const dim3 block(UTIL_BLOCK_WIDTH, UTIL_BLOCK_HEIGHT);
    const dim3 grid(cv::gpu::divUp(image.cols, block.x), cv::gpu::divUp(image.rows, block.y));
    scaledSet16SC1Mask16SC1<<<grid, block, 0, st>>>(image.data, image.rows, image.cols, image.step, val, mask.data, mask.step);
    //cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize());
}

void scaledSet16SC1Mask32SC1(cv::gpu::GpuMat& image, short val, const cv::gpu::GpuMat& mask, cv::gpu::Stream& stream)
{
    CV_Assert(image.data && image.type() == CV_16SC1 &&
        mask.data && mask.type() == CV_32SC1 && image.size() == mask.size());
    hipStream_t st = cv::gpu::StreamAccessor::getStream(stream);
    const dim3 block(UTIL_BLOCK_WIDTH, UTIL_BLOCK_HEIGHT);
    const dim3 grid(cv::gpu::divUp(image.cols, block.x), cv::gpu::divUp(image.rows, block.y));
    scaledSet16SC1Mask32SC1<<<grid, block, 0, st>>>(image.data, image.rows, image.cols, image.step, val, mask.data, mask.step);
    //cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize());
}

void divide32SC4To16SC4(const cv::gpu::GpuMat& srcImage, const cv::gpu::GpuMat& srcAlpha, cv::gpu::GpuMat& dstImage, cv::gpu::Stream& stream)
{
    CV_Assert(srcImage.data && srcImage.type() == CV_32SC4 &&
        srcAlpha.data && srcAlpha.type() == CV_32SC1 &&
        srcImage.size() == srcAlpha.size());

    dstImage.create(srcImage.size(), CV_16SC4);

    hipStream_t st = cv::gpu::StreamAccessor::getStream(stream);
    const dim3 block(32, 8);
    const dim3 grid(cv::gpu::divUp(srcImage.cols, block.x), cv::gpu::divUp(srcImage.rows, block.y));
    divide32SC4To16SC4<<<grid, block, 0, st>>>(srcImage.data, srcImage.rows, srcImage.cols, srcImage.step,
        srcAlpha.data, srcAlpha.rows, srcAlpha.cols, srcAlpha.step,
        dstImage.data, dstImage.rows, dstImage.cols, dstImage.step);
    //cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize());
}

void subtract16SC4(const cv::gpu::GpuMat& a, const cv::gpu::GpuMat& b, cv::gpu::GpuMat& c, cv::gpu::Stream& stream)
{
    CV_Assert(a.data && a.type() == CV_16SC4 &&
        b.data && b.type() == CV_16SC4 && a.size() == b.size());

    c.create(a.size(), CV_16SC4);

    hipStream_t st = cv::gpu::StreamAccessor::getStream(stream);
    const dim3 block(UTIL_BLOCK_WIDTH, UTIL_BLOCK_HEIGHT);
    const dim3 grid(cv::gpu::divUp(a.cols, block.x), cv::gpu::divUp(a.rows, block.y));
    subtract16SC4<<<grid, block, 0, st>>>(a.data, a.step, b.data, b.step, c.data, c.step, a.rows, a.cols);
    //cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize());
}

void add32SC4(const cv::gpu::GpuMat& a, const cv::gpu::GpuMat& b, cv::gpu::GpuMat& c, cv::gpu::Stream& stream)
{
    CV_Assert(a.data && a.type() == CV_32SC4 &&
        b.data && b.type() == CV_32SC4 && a.size() == b.size());

    c.create(a.size(), CV_32SC4);

    hipStream_t st = cv::gpu::StreamAccessor::getStream(stream);
    const dim3 block(UTIL_BLOCK_WIDTH, UTIL_BLOCK_HEIGHT);
    const dim3 grid(cv::gpu::divUp(a.cols, block.x), cv::gpu::divUp(a.rows, block.y));
    add32SC4<<<grid, block, 0, st>>>(a.data, a.step, b.data, b.step, c.data, c.step, a.rows, a.cols);
    //cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize());
}

__global__ void func(const unsigned char* src, int srcStep,
    unsigned char* dst, int dstStep, int rows, int cols)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x < cols && y < rows)
    {
        float val = getElem<float>(src, srcStep, y, x);
        val = val * val * val;
        val = val * val;
        val = val - val * val * val;
        getRowPtr<float>(dst, dstStep, y)[x] = val;
    }
}

void func(const cv::gpu::GpuMat& src, cv::gpu::GpuMat& dst)
{
    CV_Assert(src.data && src.type() == CV_32FC1);
    dst.create(src.size(), src.type());
    const dim3 block(UTIL_BLOCK_WIDTH, UTIL_BLOCK_HEIGHT);
    const dim3 grid(cv::gpu::divUp(src.cols, block.x), cv::gpu::divUp(src.rows, block.y));
    func<<<grid, block>>>(src.data, src.step, dst.data, dst.step, src.rows, src.cols);
    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipDeviceSynchronize());
}