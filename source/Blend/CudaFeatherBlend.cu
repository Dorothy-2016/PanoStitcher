#include "hip/hip_runtime.h"
#include "opencv2/core.hpp"
#include "opencv2/core/cuda.hpp"
#include "opencv2/core/cuda/common.hpp"
#include "CudaUtil.cuh"
#include "hip/hip_runtime.h"
#include ""

__global__ void accumulate8UC4To32SC4(const unsigned char* srcData, int srcRows, int srcCols, int srcStep,
    const unsigned char* weightData, int weightRows, int weightCols, int weightStep,
    unsigned char* dstData, int dstRows, int dstCols, int dstStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < srcCols && y < srcRows)
    {
        getRowPtr<int4>(dstData, dstStep, y)[x] = getRowPtr<int4>(dstData, dstStep, y)[x] + 
            getElem<int>(weightData, weightStep, y, x) * getElem<uchar4>(srcData, srcStep, y, x);
    }
}

__global__ void normalize32SC4Feather(unsigned char* imageData, int imageRows, int imageCols, int imageStep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < imageCols && y < imageRows)
    {
        getRowPtr<int4>(imageData, imageStep, y)[x] = getElem<int4>(imageData, imageStep, y, x) >> 16;
    }
}

void accumulate8UC4To32SC4(const cv::cuda::GpuMat& src, const cv::cuda::GpuMat& weight, cv::cuda::GpuMat& dst)
{
    CV_Assert(src.data && src.type() == CV_8UC4 &&
        weight.data && weight.type() == CV_32SC1 &&
        dst.data && dst.type() == CV_32SC4 &&
        src.size() == weight.size() && src.size() == dst.size());

    const dim3 block(32, 8);
    const dim3 grid(cv::cuda::device::divUp(src.cols, block.x), cv::cuda::device::divUp(src.rows, block.y));
    accumulate8UC4To32SC4<<<grid, block>>>(src.data, src.rows, src.cols, src.step, 
        weight.data, weight.rows, weight.cols, weight.step,
        dst.data, dst.rows, dst.cols, dst.step);
    //cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize());
}

void normalize32SC4Feather(cv::cuda::GpuMat& image)
{
    CV_Assert(image.data && image.type() == CV_32SC4);
    const dim3 block(32, 8);
    const dim3 grid(cv::cuda::device::divUp(image.cols, block.x), cv::cuda::device::divUp(image.rows, block.y));
    normalize32SC4Feather<<<grid, block>>>(image.data, image.rows, image.cols, image.step);
    //cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize());
}