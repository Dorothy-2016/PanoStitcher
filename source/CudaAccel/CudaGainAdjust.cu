#include "hip/hip_runtime.h"
#include "opencv2/core.hpp"
#include "opencv2/core/cuda.hpp"
#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda_stream_accessor.hpp"
#include "CudaUtil.cuh"
#include "hip/hip_runtime.h"
#include ""

__constant__ unsigned char cudaLUT[256];

__global__ void transformKernel8UC4(const unsigned char* srcData, int srcStep,
    unsigned char* dstData, int dstStep, unsigned char* maskData, int maskStep, int rows, int cols)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < cols && y < rows)
    {
        const unsigned char* srcPtr = getRowPtr<unsigned char>(srcData, srcStep, y) + x * 4;
        unsigned char* dstPtr = getRowPtr<unsigned char>(dstData, dstStep, y) + x * 4;
        if (!maskData || (maskData && getElem<unsigned char>(maskData, maskStep, y, x)))
        {
            dstPtr[0] = cudaLUT[srcPtr[0]];
            dstPtr[1] = cudaLUT[srcPtr[1]];
            dstPtr[2] = cudaLUT[srcPtr[2]];
            dstPtr[3] = 0;
        }
        else
        {
            dstPtr[0] = 0;
            dstPtr[1] = 0;
            dstPtr[2] = 0;
            dstPtr[3] = 0;
        }
    }
}

void cudaTransform(const cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst, 
    const std::vector<unsigned char>& lut, cv::cuda::Stream& stream, const cv::cuda::GpuMat& mask)
{
    CV_Assert(src.data && src.type() == CV_8UC4 && lut.size() == 256);
    CV_Assert((mask.data && mask.type() == CV_8UC1 && mask.size() == src.size()) || !mask.data);

    int rows = src.rows, cols = src.cols;
    dst.create(rows, cols, CV_8UC4);

    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cudaLUT), lut.data(), 256));

    dim3 block(16, 16);
    dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
    hipStream_t st = cv::cuda::StreamAccessor::getStream(stream);
    transformKernel8UC4<<<grid, block, 0, st>>>(src.data, src.step, dst.data, dst.step, mask.data, mask.step, rows, cols);
    cudaSafeCall(hipGetLastError());
}

__constant__ unsigned char cudaLUTB[256], cudaLUTG[256], cudaLUTR[256];

__global__ void transformBGRKernel8UC4(const unsigned char* srcData, int srcStep,
    unsigned char* dstData, int dstStep, unsigned char* maskData, int maskStep, int rows, int cols)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < cols && y < rows)
    {
        const unsigned char* srcPtr = getRowPtr<unsigned char>(srcData, srcStep, y) + x * 4;
        unsigned char* dstPtr = getRowPtr<unsigned char>(dstData, dstStep, y) + x * 4;
        if (!maskData || (maskData && getElem<unsigned char>(maskData, maskStep, y, x)))
        {
            dstPtr[0] = cudaLUTB[srcPtr[0]];
            dstPtr[1] = cudaLUTG[srcPtr[1]];
            dstPtr[2] = cudaLUTR[srcPtr[2]];
            dstPtr[3] = 0;
        }
        else
        {
            dstPtr[0] = 0;
            dstPtr[1] = 0;
            dstPtr[2] = 0;
            dstPtr[3] = 0;
        }
    }
}

void cudaTransform(const cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst, 
    const std::vector<std::vector<unsigned char> >& luts, cv::cuda::Stream& stream, const cv::cuda::GpuMat& mask)
{
    CV_Assert(src.data && src.type() == CV_8UC4 && 
        luts.size() == 3 && luts[0].size() == 256 && luts[1].size() == 256 && luts[2].size() == 256);
    CV_Assert((mask.data && mask.type() == CV_8UC1 && mask.size() == src.size()) || !mask.data);

    int rows = src.rows, cols = src.cols;
    dst.create(rows, cols, CV_8UC4);

    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cudaLUTB), luts[0].data(), 256));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cudaLUTG), luts[1].data(), 256));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cudaLUTR), luts[2].data(), 256));

    dim3 block(16, 16);
    dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
    hipStream_t st = cv::cuda::StreamAccessor::getStream(stream);
    transformBGRKernel8UC4<<<grid, block, 0, st>>>(src.data, src.step, dst.data, dst.step, mask.data, mask.step, rows, cols);
    cudaSafeCall(hipGetLastError());
}