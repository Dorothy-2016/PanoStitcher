#include "hip/hip_runtime.h"
#include "opencv2/core.hpp"
#include "opencv2/core/cuda.hpp"
#include "opencv2/core/cuda_stream_accessor.hpp"
#include "opencv2/core/cuda/common.hpp"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "ZReproject.h"

typedef double CalcType;

struct CudaRemapParam
{
    enum ImageType
    {
        ImageTypeRectlinear = 0,
        ImageTypeFullFrameFishEye = 1,
        ImageTypeDrumFishEye = 2,
        ImageTypeCircularFishEye = 3
    };
    CalcType srcTX, srcTY;
    CalcType destTX, destTY;
    CalcType scale[2];
    CalcType shear[2];
    CalcType rot[2];
    void *perspect[2];
    CalcType rad[6];
    CalcType mt[3][3];
    CalcType distance;
    CalcType horizontal;
    CalcType vertical;
    CalcType PI;
    CalcType width;
    CalcType height;
    CalcType centx;
    CalcType centy;
    CalcType sqrDist;
    int imageType;
};

void copyParam(const Remap& src, CudaRemapParam& dst, 
    CalcType width, CalcType height, CalcType centx, CalcType centy, CalcType sqrDist, int type)
{
    dst.srcTX = src.srcTX;
    dst.srcTY = src.srcTY;
    dst.destTX = src.destTX;
    dst.destTY = src.destTY;
    dst.scale[0] = src.mp.scale[0];
    dst.scale[1] = src.mp.scale[1];
    dst.shear[0] = src.mp.shear[0];
    dst.shear[1] = src.mp.shear[1];
    dst.rot[0] = src.mp.rot[0];
    dst.rot[1] = src.mp.rot[1];
    dst.perspect[0] = src.mp.perspect[0];
    dst.perspect[1] = src.mp.perspect[1];
    for (int i = 0; i < 6; i++)
        dst.rad[i] = src.mp.rad[i];
    for (int i = 0; i < 3; i++)
    {
        for (int j = 0; j < 3; j++)
            dst.mt[i][j] = src.mp.mt[i][j];
    }    
    dst.distance = src.mp.distance;
    dst.horizontal = src.mp.horizontal;
    dst.vertical = src.mp.vertical;
    dst.PI = 3.1415926535898;
    dst.width = width;
    dst.height = height;
    dst.centx = centx;
    dst.centy = centy;
    dst.sqrDist = sqrDist;
    dst.imageType = type;
}

__constant__ CudaRemapParam param;

__device__ void dstToSrc(float* srcx, float* srcy, int dstx, int dsty, int mapWidth, int mapHeight)
{
    if (dstx >= mapWidth || dsty >= mapHeight)
        return;

    CalcType x_src = dstx, y_src = dsty;

    x_src -= param.srcTX - 0.5;
    y_src -= param.srcTY - 0.5;

    CalcType tx_dest, ty_dest;

    //rotate_erect  ���Ĺ�һ��
    tx_dest = x_src + param.rot[1];

    while (tx_dest < -param.rot[0])
        tx_dest += 2 * param.rot[0];

    while (tx_dest >   param.rot[0])
        tx_dest -= 2 * param.rot[0];

    ty_dest = y_src;

    x_src = tx_dest;
    y_src = ty_dest;

    //sphere_tp_erect ��������ת��Ϊ��ʵ����
    CalcType phi, theta, r;
    CalcType v[3];
    phi = x_src / param.distance; //
    theta = -y_src / param.distance + param.PI / 2; //
    if (theta < 0)
    {
        theta = -theta;
        phi += param.PI;
    }
    if (theta > param.PI)
    {
        theta = param.PI - (theta - param.PI);
        phi += param.PI;
    }

    v[0] = sin(theta) * sin(phi);
    v[1] = cos(theta);
    v[2] = sin(theta) * cos(phi);

    //��������
    CalcType v0 = v[0];
    CalcType v1 = v[1];
    CalcType v2 = v[2];

    for (int i = 0; i<3; i++)
    {
        v[i] = param.mt[0][i] * v0 + param.mt[1][i] * v1 + param.mt[2][i] * v2;
    }

    r = sqrt(v[0] * v[0] + v[1] * v[1]);
    if (r == 0.0)
        theta = 0.0;
    else
        theta = param.distance * atan2(r, v[2]) / r;
    tx_dest = theta * v[0];
    ty_dest = theta * v[1];
    x_src = tx_dest;
    y_src = ty_dest;

    if (param.imageType == CudaRemapParam::ImageTypeRectlinear)                                    // rectilinear image
    {
        //SetDesc(m_stack[i],   rect_sphere_tp,         &(m_mp.distance) ); i++; // Convert rectilinear to spherical
        CalcType rho, theta, r;
        r = sqrt(x_src * x_src + y_src * y_src);
        theta = r / param.distance;

        if (theta >= param.PI / 2.0)
            rho = 1.6e16;
        else if (theta == 0.0)
            rho = 1.0;
        else
            rho = tan(theta) / theta;
        tx_dest = rho * x_src;
        ty_dest = rho * y_src;
        x_src = tx_dest;
        y_src = ty_dest;
    }

    //������ڲ�
    //SetDesc(  stack[i],   resize,                 param.scale       ); i++; // Scale image
    tx_dest = x_src * param.scale[0];
    ty_dest = y_src * param.scale[1];

    x_src = tx_dest;
    y_src = ty_dest;

    CalcType rt, scale;

    rt = (sqrt(x_src*x_src + y_src*y_src)) / param.rad[4];
    if (rt < param.rad[5])
    {
        scale = ((param.rad[3] * rt + param.rad[2]) * rt +
            param.rad[1]) * rt + param.rad[0];
    }
    else
        scale = 1000.0;

    tx_dest = x_src * scale;
    ty_dest = y_src * scale;

    x_src = tx_dest;
    y_src = ty_dest;

    //�����ˮƽ��ֱ����
    if (param.vertical != 0.0)
    {
        //SetDesc(stack[i],   vert,                   &(param.vertical));   i++;
        tx_dest = x_src;
        ty_dest = y_src + param.vertical;
        x_src = tx_dest;
        y_src = ty_dest;
    }

    if (param.horizontal != 0.0)
    {
        //SetDesc(stack[i],   horiz,                  &(param.horizontal)); i++;
        tx_dest = x_src + param.horizontal;
        ty_dest = y_src;
        x_src = tx_dest;
        y_src = ty_dest;
    }

    if (param.shear[0] != 0 || param.shear[1] != 0)
    {
        //SetDesc( stack[i],  shear,                  param.shear       ); i++;
        tx_dest = x_src + param.shear[0] * y_src;
        ty_dest = y_src + param.shear[1] * x_src;
    }

    tx_dest += param.destTX - 0.5;
    ty_dest += param.destTY - 0.5;

    if (param.imageType == CudaRemapParam::ImageTypeDrumFishEye ||
        param.imageType == CudaRemapParam::ImageTypeCircularFishEye)
    {
        float diffx = tx_dest - param.centx;
        float diffy = ty_dest - param.centy;
        if (tx_dest >= 0 && tx_dest < param.width && ty_dest >= 0 && ty_dest < param.height &&
            diffx * diffx + diffy * diffy < param.sqrDist)
        {
            *srcx = tx_dest;
            *srcy = ty_dest;
        }
        else
        {
            *srcx = -1.0F;
            *srcy = -1.0F;
        }
    }
    else
    {
        *srcx = tx_dest;
        *srcy = ty_dest;
    }
}

__global__ void remapKernel(unsigned char* xMapData, int xMapStep, 
    unsigned char* yMapData, int yMapStep, int mapWidth, int mapHeight)
{
    
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= mapWidth || y >= mapHeight)
        return;

    CalcType x_src = x, y_src = y;

    x_src -= param.srcTX - 0.5;
    y_src -= param.srcTY - 0.5;

    CalcType tx_dest, ty_dest;

    //rotate_erect  ���Ĺ�һ��
    tx_dest = x_src + param.rot[1];

    while (tx_dest < -param.rot[0])
        tx_dest += 2 * param.rot[0];

    while (tx_dest >   param.rot[0])
        tx_dest -= 2 * param.rot[0];

    ty_dest = y_src;

    x_src = tx_dest;
    y_src = ty_dest;

    //sphere_tp_erect ��������ת��Ϊ��ʵ����
    CalcType phi, theta, r;
    CalcType v[3];
    phi = x_src / param.distance; //
    theta = -y_src / param.distance + param.PI / 2; //
    if (theta < 0)
    {
        theta = -theta;
        phi += param.PI;
    }
    if (theta > param.PI)
    {
        theta = param.PI - (theta - param.PI);
        phi += param.PI;
    }

    v[0] = sin(theta) * sin(phi);
    v[1] = cos(theta);
    v[2] = sin(theta) * cos(phi);

    //��������
    CalcType v0 = v[0];
    CalcType v1 = v[1];
    CalcType v2 = v[2];

    for (int i = 0; i<3; i++)
    {
        v[i] = param.mt[0][i] * v0 + param.mt[1][i] * v1 + param.mt[2][i] * v2;
    }

    r = sqrt(v[0] * v[0] + v[1] * v[1]);
    if (r == 0.0)
        theta = 0.0;
    else
        theta = param.distance * atan2(r, v[2]) / r;
    tx_dest = theta * v[0];
    ty_dest = theta * v[1];
    x_src = tx_dest;
    y_src = ty_dest;

    if (param.imageType == CudaRemapParam::ImageTypeRectlinear)                                    // rectilinear image
    {
        //SetDesc(m_stack[i],   rect_sphere_tp,         &(m_mp.distance) ); i++; // Convert rectilinear to spherical
        CalcType rho, theta, r;
        r = sqrt(x_src * x_src + y_src * y_src);
        theta = r / param.distance;

        if (theta >= param.PI / 2.0)
            rho = 1.6e16;
        else if (theta == 0.0)
            rho = 1.0;
        else
            rho = tan(theta) / theta;
        tx_dest = rho * x_src;
        ty_dest = rho * y_src;
        x_src = tx_dest;
        y_src = ty_dest;
    }

    //������ڲ�
    //SetDesc(  stack[i],   resize,                 param.scale       ); i++; // Scale image
    tx_dest = x_src * param.scale[0];
    ty_dest = y_src * param.scale[1];

    x_src = tx_dest;
    y_src = ty_dest;

    CalcType rt, scale;

    rt = (sqrt(x_src*x_src + y_src*y_src)) / param.rad[4];
    if (rt < param.rad[5])
    {
        scale = ((param.rad[3] * rt + param.rad[2]) * rt +
            param.rad[1]) * rt + param.rad[0];
    }
    else
        scale = 1000.0;

    tx_dest = x_src * scale;
    ty_dest = y_src * scale;

    x_src = tx_dest;
    y_src = ty_dest;

    //�����ˮƽ��ֱ����
    if (param.vertical != 0.0)
    {
        //SetDesc(stack[i],   vert,                   &(param.vertical));   i++;
        tx_dest = x_src;
        ty_dest = y_src + param.vertical;
        x_src = tx_dest;
        y_src = ty_dest;
    }

    if (param.horizontal != 0.0)
    {
        //SetDesc(stack[i],   horiz,                  &(param.horizontal)); i++;
        tx_dest = x_src + param.horizontal;
        ty_dest = y_src;
        x_src = tx_dest;
        y_src = ty_dest;
    }

    if (param.shear[0] != 0 || param.shear[1] != 0)
    {
        //SetDesc( stack[i],  shear,                  param.shear       ); i++;
        tx_dest = x_src + param.shear[0] * y_src;
        ty_dest = y_src + param.shear[1] * x_src;
    }

    tx_dest += param.destTX - 0.5;
    ty_dest += param.destTY - 0.5;

    if (param.imageType == CudaRemapParam::ImageTypeDrumFishEye || 
        param.imageType == CudaRemapParam::ImageTypeCircularFishEye)
    {
        float diffx = tx_dest - param.centx;
        float diffy = ty_dest - param.centy;
        if (tx_dest >= 0 && tx_dest < param.width && ty_dest >= 0 && ty_dest < param.height &&
            diffx * diffx + diffy * diffy < param.sqrDist)
        {
            *((float*)(xMapData + y * xMapStep) + x) = tx_dest;
            *((float*)(yMapData + y * yMapStep) + x) = ty_dest;
        }
        else
        {
            *((float*)(xMapData + y * xMapStep) + x) = -1.0F;
            *((float*)(yMapData + y * yMapStep) + x) = -1.0F;
        }
    }
    else
    {
        *((float*)(xMapData + y * xMapStep) + x) = tx_dest;
        *((float*)(yMapData + y * yMapStep) + x) = ty_dest;
    }
    
    //int x = threadIdx.x + blockIdx.x * blockDim.x;
    //int y = threadIdx.y + blockIdx.y * blockDim.y;
    //if (x >= mapWidth || y >= mapHeight)
    //    return;

    //dstToSrc((float*)(xMapData + y * xMapStep) + x, (float*)(yMapData + y * yMapStep) + x, x, y, mapWidth, mapHeight);
}

void cudaGenerateReprojectMap(const PhotoParam& photoParam_,
    const cv::Size& srcSize, const cv::Size& dstSize, cv::cuda::GpuMat& xmap, cv::cuda::GpuMat& ymap)
{
    CV_Assert(srcSize.width > 0 && srcSize.height > 0 &&
        dstSize.width > 0 && dstSize.height > 0 && dstSize.width == 2 * dstSize.height);

    int dstWidth = dstSize.width, dstHeight = dstSize.height;
    int srcWidth = srcSize.width, srcHeight = srcSize.height;

    bool fullImage = (photoParam_.imageType == PhotoParam::ImageTypeRectlinear) || 
                     (photoParam_.imageType == PhotoParam::ImageTypeFullFrameFishEye);
    PhotoParam photoParam = photoParam_;
    if (fullImage)
    {
        photoParam.cropX = 0;
        photoParam.cropY = 0;
        photoParam.cropWidth = dstWidth;
        photoParam.cropHeight = dstHeight;
    }
    CalcType centx = 0, centy = 0, sqrDist = 0;
    if (photoParam.circleR == 0)
    {
        centx = photoParam.cropX + photoParam.cropWidth / 2;
        centy = photoParam.cropY + photoParam.cropHeight / 2;
        sqrDist = photoParam.cropWidth > photoParam.cropHeight ?
            photoParam.cropWidth * photoParam.cropWidth * 0.25 :
            photoParam.cropHeight * photoParam.cropHeight * 0.25;
    }
    else
    {
        centx = photoParam.circleX;
        centy = photoParam.circleY;
        sqrDist = photoParam.circleR * photoParam.circleR;
    }

    Remap remap;
    remap.init(photoParam, dstWidth, dstHeight, srcWidth, srcHeight);
    CudaRemapParam cudaParam;
    copyParam(remap, cudaParam, srcWidth, srcHeight, centx, centy, sqrDist, photoParam.imageType);
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(param), &cudaParam, sizeof(CudaRemapParam)));

    xmap.create(dstHeight, dstWidth, CV_32FC1);
    ymap.create(dstHeight, dstWidth, CV_32FC1);

    dim3 block(16, 16);
    dim3 grid((dstSize.width + block.x - 1) / block.x, (dstSize.height + block.y - 1) / block.y);
    remapKernel<<<grid, block>>>(xmap.data, xmap.step, ymap.data, ymap.step, dstWidth, dstHeight);
    cudaSafeCall(hipGetLastError());
}

void cudaGenerateReprojectMaps(const std::vector<PhotoParam>& params,
    const cv::Size& srcSize, const cv::Size& dstSize, std::vector<cv::cuda::GpuMat>& xmaps, std::vector<cv::cuda::GpuMat>& ymaps)
{
    int num = params.size();
    xmaps.resize(num);
    ymaps.resize(num);
    for (int i = 0; i < num; i++)
        cudaGenerateReprojectMap(params[i], srcSize, dstSize, xmaps[i], ymaps[i]);
}